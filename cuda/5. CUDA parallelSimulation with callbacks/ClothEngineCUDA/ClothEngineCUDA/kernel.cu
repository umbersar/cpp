#include "hip/hip_runtime.h"
#include "Python.h"
#include "hip/hip_runtime.h"
#include ""
#include "arrayobject.h"
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <omp.h>


#define IDATA(p) ((int *) (((PyArrayObject *)p)->data))
#define DDATA(p) ((double *) (((PyArrayObject *)p)->data))

void c_compute_force(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
	double *xPos, double *yPos, double *zPos);

static PyObject *py_cFullEngine(PyObject *self, PyObject *args);
void c_full_engine(int N,int interact,double myBallX,double myBallY,double myBallZ, double myBallRadius, double separation, double mass,double fcon,
	double gravity,double ballsize,double offset,double dt,int update,  void (*renderui)(double** nodepos, int N), int threads);
PyObject* c_full_engine_Cudafied(int N,int interact,double myBallX,double myBallY,double myBallZ, double myBallRadius, double separation, double mass,double fcon,
	double gravity,double ballsize,double offset,double dt,int update,  void (*renderui)(double** nodepos, int N), int threads);


__device__ int maxOnDevice(int a, int b) {
	if (a > b) {
		return a;
	} else {
		return b;
	}
}//end max

__device__ int minOnDevice(int a, int b) {
	if (a > b) {
		return b;
	} else {
		return a;
	}
}//end min

__device__ double magOnDevice(double x, double y, double z) {
	return sqrt(x*x + y*y +z*z);
}//end min

__device__ double normxOnDevice(double x, double y, double z) {
	return x/sqrt(x*x + y*y +z*z);
}//end min

__device__ double normyOnDevice(double x, double y, double z) {
	return y/sqrt(x*x + y*y +z*z);
}//end min

__device__ double normzOnDevice(double x, double y, double z) {
	return z/sqrt(x*x + y*y +z*z);
}//end min

int maxOnHost(int a, int b) {
	if (a > b) {
		return a;
	} else {
		return b;
	}
}//end max

int minOnHost(int a, int b) {
	if (a > b) {
		return b;
	} else {
		return a;
	}
}//end min


void GetPosInXYZDirection(double** nodepos, double *xPos, double *yPos, double *zPos, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
		xPos[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		yPos[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		zPos[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xPos[i] = nodepos[i][0];
		yPos[i] = nodepos[i][1];
		zPos[i] = nodepos[i][2];
	}
}

void GetForceInXYZDirection(double** force, double *xForce, double *yForce, double *zForce, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
		xForce[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		yForce[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		zForce[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xForce[i] = force[i][0];
		yForce[i] = force[i][1];
		zForce[i] = force[i][2];
	}
}

void GetVelInXYZDirection(double** velocity, double *xVel, double *yVel, double *zVel, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
		xVel[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		yVel[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		zVel[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xVel[i] = velocity[i][0];
		yVel[i] = velocity[i][1];
		zVel[i] = velocity[i][2];
	}
}

//this function is similiar to create_cloth as it initializes the data structures used for simulation
void Initialize(int N,double separation, double offset, double ballsize, double** velocity, double** force, double** oldforce, double** nodepos)
{		
	//This is the conventional way to access two dim array. 
	//initialize the two dim array(matrix) to 
	// for(int i = 0; i < N*N; i++)
	// for(int j = 0; j < 3; j++)
	// velocity[i][j] = 0.0;

	//this is same but a bit absurd way of doing same.
	// for(int nx=0;i<N;i++)
	// {
	// for(int ny=0;i<N;i++)
	// {
	// velocity[nx*N+ny][0] = 0.0;
	// }
	// }

	for(int nx=0;nx<N;nx++)
	{
		double x = nx*separation-(N-1)*separation*0.5+offset;
		for(int ny=0;ny<N;ny++)
		{
			double y = ny*separation-(N-1)*separation*0.5+offset;

			nodepos[nx*N+ny][0] = x;
			nodepos[nx*N+ny][1] = ballsize+1.0;
			nodepos[nx*N+ny][2] = y;

			velocity[nx*N+ny][0] = 0.0;
			velocity[nx*N+ny][1] = 0.0;
			velocity[nx*N+ny][2] = 0.0;

			force[nx*N+ny][0] = 0.0;
			force[nx*N+ny][1] = 0.0;
			force[nx*N+ny][2] = 0.0;

			oldforce[nx*N+ny][0] = 0.0;
			oldforce[nx*N+ny][1] = 0.0;
			oldforce[nx*N+ny][2] = 0.0;
		}
	}
}

//int max(int a, int b) {
//	if (a > b) {
//		return a;
//	} else {
//		return b;
//	}
//}//end max
//
//int min(int a, int b) {
//	if (a > b) {
//		return b;
//	} else {
//		return a;
//	}
//}//end min

double mag(double x, double y, double z) {
	return sqrt(x*x + y*y +z*z);
}//end min

double normx(double x, double y, double z) {
	return x/sqrt(x*x + y*y +z*z);
}//end min

double normy(double x, double y, double z) {
	return y/sqrt(x*x + y*y +z*z);
}//end min

double normz(double x, double y, double z) {
	return z/sqrt(x*x + y*y +z*z);
}//end min

static PyObject *py_renderui_func = NULL;

static void stub_renderui_func(double** nodepos, int N)
{
	/*int testargument=121;
	PyObject *arglist = Py_BuildValue("(i)", testargument);*/
	PyObject *pos_of_each_ball_list = Py_BuildValue("[]");
	if (!pos_of_each_ball_list)
		printf("there is an error");
	for (int nx=0; nx<N; nx++)
	{
		for (int ny=0; ny<N; ny++)
		{
			PyObject *lc = Py_BuildValue("(ddd)",nodepos[nx*N+ny][0],nodepos[nx*N+ny][1],nodepos[nx*N+ny][2]);
			PyList_Append(pos_of_each_ball_list,lc);
			Py_DECREF(lc);
		}
	}
	PyObject *arglist = Py_BuildValue("(O)",pos_of_each_ball_list);
	
	// ...for calling the Python rendering function.
	PyObject *result = PyEval_CallObject(py_renderui_func,arglist);

	Py_DECREF(arglist);
}

void c_compute_force(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
double *xPos, double *yPos, double *zPos){

	// double r12X =0.0;
	// double r12Y =0.0;
	// double r12Z =0.0;
	//r12=vector(0.0,0.0,0.0)
	int nx,ny,dx,dy;
	//double PE=0.0;
	//double len=0.0;

	#pragma omp parallel for default(none) shared(N,delta,gravity,separation,fcon,xForce,yForce,zForce,xPos,yPos,zPos) private(nx,ny,dx,dy)	
	for (nx=0; nx<N; nx++)
	{
		for (ny=0; ny<N; ny++)
		{
			xForce[nx*N+ny] = 0.0;
			yForce[nx*N+ny] = -gravity;
			zForce[nx*N+ny] = 0.0;

			int lowerValuedx = max(nx-delta,0);
			int upperValuedx=min(nx+delta+1,N);
			for(dx=lowerValuedx; dx<upperValuedx;dx++)
			{
				int lowerValuedy=max(ny-delta,0);
				int upperValuedy=min(ny+delta+1,N);
				for(dy=lowerValuedy; dy<upperValuedy;dy++)
				{
					double len=sqrt((double)((nx-dx)*(nx-dx)+(ny-dy)*(ny-dy)) ) *separation;

					if (nx!=dx || ny!=dy)
					{
						double r12X = xPos[dx*N+dy] - xPos[nx*N+ny];
						double r12Y = yPos[dx*N+dy] - yPos[nx*N+ny];
						double r12Z = zPos[dx*N+dy] - zPos[nx*N+ny];
						//PE = PE + fcon*(mag(r12X,r12Y,r12Z)-len)*(mag(r12X,r12Y,r12Z)-len);
						xForce[nx*N+ny] = xForce[nx*N+ny] +fcon*normx(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
						yForce[nx*N+ny]= yForce[nx*N+ny] +fcon*normy(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
						zForce[nx*N+ny]= zForce[nx*N+ny] +fcon*normz(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
					}
				}
			}

		}
	}
}

//ramneek: trying to cuda'fy this code
__global__ void MyKernel(int *Nptr,int *deltaptr, double *gravityptr, double *separationptr, double *fconptr, double *xForce, double *yForce, double *zForce,
	double *xPos, double *yPos, double *zPos/*, PyObject *force_on_each_ball_list*/ )
{
	int N = *Nptr;
	//*Nptr =43332;
	int delta= *deltaptr;
	double gravity= *gravityptr;
	double separation = *separationptr;
	double fcon = *fconptr;

	double len=0.0;
	double r12X =0.0;
	double r12Y =0.0;
	double r12Z =0.0;
	double PE=0.0;


	int nx = blockDim.x * blockIdx.x + threadIdx.x;//use this place of nx
	//int ny = blockDim.x * blockIdx.x + threadIdx.y;//use this place of ny
	int ny = blockDim.y * blockIdx.y + threadIdx.y;
	//printf("nx:%d ny:%d\n", nx,ny);

	if(!(nx< N && ny <N))
		return;
	//printf("nx:%d ny:%d\n", nx,ny);


	xForce[nx*N+ny] = 0.0;
	yForce[nx*N+ny] = -gravity;
	zForce[nx*N+ny] = 0.0;

	int lowerValuedx = maxOnDevice(nx-delta,0);
	int upperValuedx=minOnDevice(nx+delta+1,N);
	for(int dx=lowerValuedx; dx<upperValuedx;dx++)
	{
		int lowerValuedy=maxOnDevice(ny-delta,0);
		int upperValuedy=minOnDevice(ny+delta+1,N);
		for(int dy=lowerValuedy; dy<upperValuedy;dy++)
		{
			len=sqrt((double)((nx-dx)*(nx-dx)+(ny-dy)*(ny-dy)) ) *separation;
			bool condition = ny!=dy;
			bool condition1 = nx!=dx;

			//if (nx!=dx || ny!=dy)
			if (condition || condition1)
			{
				r12X = xPos[dx*N+dy] - xPos[nx*N+ny];
				r12Y = yPos[dx*N+dy] - yPos[nx*N+ny];
				r12Z = zPos[dx*N+dy] - zPos[nx*N+ny];
				//PE = PE + fcon*(magOnDevice(r12X,r12Y,r12Z)-len)*(magOnDevice(r12X,r12Y,r12Z)-len);
				xForce[nx*N+ny] = xForce[nx*N+ny] +fcon*normxOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);
				yForce[nx*N+ny]= yForce[nx*N+ny] +fcon*normyOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);
				zForce[nx*N+ny]= zForce[nx*N+ny] +fcon*normzOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);

				//i tried to first get the item and modify it and set it back.
				//but then i thought....why not directly set the new item that position if it is anyways going to overwrite it.
				//PyObject *temp=PyList_GetItem(force_on_each_ball_list, nx*N+ny);
				/*ok = PyArg_ParseTuple(temp,"ddd",&N,&delta,&gravity,&separation,&fcon,&xforcearray,&yforcearray,&zforcearray,
				&xposarray,&yposarray,&zposarray);*/

				//ramneek: get the items out of xForce, yForce and zForce in the host method and use the follwing statements there. 
				/*PyObject *item = Py_BuildValue("(ddd)",xForce[nx*N+ny],yForce[nx*N+ny],zForce[nx*N+ny]);
				PyList_SetItem(force_on_each_ball_list, nx*N+ny, item);*/
			}
		}
	}
	/*for (int k=0;k<256;k++)
	{
		xForce[k] = (double)444.7;
	}*/

	/*   int i = threadIdx.x;
    c[i] = a[i] + b[i];*/
}

void c_compute_force_Cudafied(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
	double *xPos, double *yPos, double *zPos){

		/*double r12X =0.0;
		double r12Y =0.0;
		double r12Z =0.0;*/
		//r12=vector(0.0,0.0,0.0)
		int nx,ny,dx,dy;
		/*double PE=0.0;
		double len=0.0;*/




		//allocate memory on device here
		int *dev_N =0;
		int *dev_delta=0;
		double *dev_gravity=0;
		double *dev_separation=0;
		double *dev_fcon=0;

		double *dev_xForce = 0;
		double *dev_yForce = 0;
		double *dev_zForce = 0;
		double *dev_xPos = 0;
		double *dev_yPos = 0;
		double *dev_zPos = 0;

		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			exit(0);
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_N, sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_delta, sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_gravity, sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_separation, sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_fcon, sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		// Allocate GPU buffers for 6 vectors    .
		cudaStatus = hipMalloc((void**)&dev_xForce, N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_yForce,  N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_zForce,  N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_xPos, N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_yPos,  N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_zPos,  N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}


		cudaStatus = hipMemcpy(dev_N, &N,  sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_delta, &delta,  sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_gravity, &gravity,  sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_separation, &separation,  sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_fcon, &fcon,  sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_xForce, xForce, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_yForce, yForce, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_zForce, zForce, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_xPos, xPos, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_yPos, yPos, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_zPos, zPos, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		//launch the kernel here
		//dim3 threadsPerBlock(2, 2); 
		dim3 threadsPerBlock(16, 16); 
		//int blocksPerGrid = (N*N + (threadsPerBlock.x*threadsPerBlock.y) - 1) / (threadsPerBlock.x*threadsPerBlock.y);
		dim3 blocksPerGrid(ceil((double)N / threadsPerBlock.x), ceil((double)N / threadsPerBlock.y));
		// Launch a kernel on the GPU with one thread for each element.
		//addKernel<<<1, size>>>(dev_c, dev_a, dev_b,size);
		//addKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_c, dev_a, dev_b,N);
 		MyKernel<<<blocksPerGrid, threadsPerBlock>>>( dev_N,dev_delta,dev_gravity,dev_separation,dev_fcon,dev_xForce,
			dev_yForce,dev_zForce,dev_xPos,dev_yPos,dev_zPos); 

		/*int numBlocks = 1; 
		dim3 threadsPerBlock(N, N); */
		/*MyKernel<<<numBlocks, threadsPerBlock>>>( dev_N,dev_delta,dev_gravity,dev_separation,dev_fcon,dev_xForce,
			dev_yForce,dev_zForce,dev_xPos,dev_yPos,dev_zPos); */

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(xForce, dev_xForce, N*N * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		//int *testptr;
		//testptr = (int*)malloc(sizeof(int));
		//// Copy output vector from GPU buffer to host memory.
		//cudaStatus = hipMemcpy(testptr, dev_N, sizeof(int), hipMemcpyDeviceToHost);
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "hipMemcpy failed!");
		//	goto Error;
		//}

		/*for (int k=0;k<256;k++)
		{
			printf(" force %E",xForce[k] );
		}*/

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(yForce, dev_yForce, N*N * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(zForce, dev_zForce, N*N * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		
Error:
		hipFree(dev_N);
		hipFree(dev_delta);
		hipFree(dev_gravity);
		hipFree(dev_separation);
		hipFree(dev_fcon);

		hipFree(dev_xForce);
		hipFree(dev_yForce);
		hipFree(dev_zForce);
		hipFree(dev_xPos);
		hipFree(dev_yPos);
		hipFree(dev_zPos);
}


static PyObject *py_cFullEngine(PyObject *self, PyObject *args){
	int N,interact,update,ok,threads;
	double myBallX,myBallY,myBallZ,myBallRadius,separation,mass,fcon,gravity,ballsize,offset,dt;


	PyObject *pyrenderuiobj;

	//cFullEngine(N,interact,myBallX,myBallY,myBallZ,myBallRadius,separation,mass,fcon,gravity,ballsize,offset,dt,update)
	ok = PyArg_ParseTuple(args,"iidddddddddddiOi",&N,&interact,&myBallX,&myBallY,&myBallZ,&myBallRadius,&separation,&mass,&fcon,
		&gravity,&ballsize,&offset,&dt,&update,&pyrenderuiobj,&threads);

	// make sure second argument is a function
	if (!PyCallable_Check(pyrenderuiobj)) {
		PyErr_SetString(PyExc_TypeError, "Need a callable object!");
	}

	py_renderui_func = pyrenderuiobj;

	if (!ok){
		fprintf(stderr,"Error (cComputeForce) in parsing arguments\n");
		exit(1);
	}

	c_full_engine(N,interact,myBallX,myBallY,myBallZ,myBallRadius,separation,mass,fcon,gravity,ballsize,offset,
		dt,update,stub_renderui_func,threads);

	//i am returning this as i think the entry functions are required to have a return type.
	PyObject *lst;
	return lst;
}

void c_full_engine(int N,int interact,double myBallX,double myBallY,double myBallZ, double myBallRadius, double separation, double mass,double fcon,
	double gravity,double ballsize,double offset,double dt,int update,  void (*renderui)(double** nodepos, int N), int threads){
	
		//printf("N=%d interact=%d ballx=%f bally%f ballz%f ballradius%f separation=%f mass=%f fcon=%f gravity=%f ballsize=%f offfset=%f dt=%f update=%d\n",
			//N,interact,myBallX,myBallY,myBallZ, myBallRadius, separation, mass,fcon,gravity,ballsize,offset,dt,update);

		//set the number of threads desired for simulation
		printf("threads are= %d\n",threads);
		printf("no. of processors are= %d\n",omp_get_num_procs());
		omp_set_num_threads(threads);

		//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
		double **velocity;
		velocity  = (double **)malloc(N * N * sizeof(double *));
		for(int i = 0; i < N * N ; i++)
			velocity[i] = (double *)malloc(3 * sizeof(double));

		//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
		double **force;
		force  = (double **)malloc(N * N * sizeof(double *));
		for(int i = 0; i < N * N ; i++)
			force[i] = (double *)malloc(3 * sizeof(double));

		//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
		double **oldforce;
		oldforce  = (double **)malloc(N * N * sizeof(double *));
		for(int i = 0; i < N * N ; i++)
			oldforce[i] = (double *)malloc(3 * sizeof(double));

		//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
		double **nodepos;
		nodepos  = (double **)malloc(N * N * sizeof(double *));
		for(int i = 0; i < N * N ; i++)
			nodepos[i] = (double *)malloc(3 * sizeof(double));

		Initialize(N,separation,offset,ballsize,velocity,force,oldforce,nodepos);

		double *xVel, *yVel,  *zVel;
		xVel = (double *)malloc(N*N*sizeof(double));
		yVel = (double *)malloc(N*N*sizeof(double));
		zVel = (double *)malloc(N*N*sizeof(double));
		GetVelInXYZDirection(velocity,xVel,yVel,zVel,N);
		//xPos, yPos, zPos = GetPosInXYZDirection(nodes)

		double *xPos, *yPos,  *zPos;
		xPos = (double *)malloc(N*N*sizeof(double));
		yPos = (double *)malloc(N*N*sizeof(double));
		zPos = (double *)malloc(N*N*sizeof(double));
		GetPosInXYZDirection(nodepos,xPos,yPos,zPos,N);
		//xPos, yPos, zPos = GetPosInXYZDirection(nodes)

		double *xForce, *yForce,  *zForce;
		xForce = (double *)malloc(N*N*sizeof(double));
		yForce = (double *)malloc(N*N*sizeof(double));
		zForce = (double *)malloc(N*N*sizeof(double));
		GetForceInXYZDirection(force,xForce,yForce,zForce,N);
		//xForce, yForce, zForce = GetForceInXYZDirection(nodes) 

		//c_compute_force(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);
		c_compute_force_Cudafied(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);


		clock_t clktime1 = clock();
		time_t RealTime1 = time(0); 

		int iter=0;
		while(1)
		{
			iter=iter+1;
			
			for(int nx=0;nx<N;nx++)
			{
				for(int ny=0;ny<N;ny++)
				{
					nodepos[nx*N+ny][0] = nodepos[nx*N+ny][0] + dt*(velocity[nx*N+ny][0]+dt*xForce[nx*N+ny]*.5);
					nodepos[nx*N+ny][1] = nodepos[nx*N+ny][1] + dt*(velocity[nx*N+ny][1]+dt*yForce[nx*N+ny]*.5);
					nodepos[nx*N+ny][2] = nodepos[nx*N+ny][2] + dt*(velocity[nx*N+ny][2]+dt*zForce[nx*N+ny]*.5);

					oldforce[nx*N+ny][0] = xForce[nx*N+ny];
					oldforce[nx*N+ny][1] = yForce[nx*N+ny];
					oldforce[nx*N+ny][2] = zForce[nx*N+ny];
				}
			}
		
			for(int i=0;i<N*N;i++)
			{
				double distX = nodepos[i][0] - myBallX;
				double distY = nodepos[i][1] - myBallY;
				double distZ = nodepos[i][2] - myBallZ;

				double dist = mag(distX,distY,distZ);
				//dist = node.pos-vector(myball.x,myball.y,myball.z)

				if(dist<myBallRadius)
				{
					// printf("%E %E %E\n",nodepos[i][0] ,nodepos[i][1],nodepos[i][2]);
					double fvectorX = (distX/dist)*myBallRadius;
					double fvectorY = (distY/dist)*myBallRadius;
					double fvectorZ = (distZ/dist)*myBallRadius;
					//fvector=dist/dist.mag*myball.radius

					nodepos[i][0] = myBallX +fvectorX;
					nodepos[i][1] = myBallY +fvectorY;
					nodepos[i][2] = myBallZ +fvectorZ;
					//node.pos=vector(myball.x,myball.y,myball.z)+fvector	

					double fvectorMag = mag(fvectorX,fvectorY,fvectorZ);
					velocity[i][0] = velocity[i][0] - (velocity[i][0]*fvectorX/fvectorMag)*(fvectorX/fvectorMag);
					velocity[i][1] = velocity[i][1] - (velocity[i][1]*fvectorY/fvectorMag)*(fvectorY/fvectorMag);
					velocity[i][2] = velocity[i][2] - (velocity[i][2]*fvectorZ/fvectorMag)*(fvectorZ/fvectorMag);
					//node.velocity = node.velocity - (dot(node.velocity,fvector/fvector.mag))*(fvector/fvector.mag)
				}
			}

			if(iter%update==0) 
			{
				renderui(nodepos,N);
			}

			GetPosInXYZDirection(nodepos,xPos,yPos,zPos,N);
			GetForceInXYZDirection(force,xForce,yForce,zForce,N);		
			//c_compute_force(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);
			c_compute_force_Cudafied(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);

			for(int nx=0;nx<N;nx++)
			{
				for(int ny=0;ny<N;ny++)
				{
					velocity[nx*N+ny][0]=velocity[nx*N+ny][0]+dt*(xForce[nx*N+ny] + oldforce[nx*N+ny][0])*0.5;
					velocity[nx*N+ny][1]=velocity[nx*N+ny][1]+dt*(yForce[nx*N+ny] + oldforce[nx*N+ny][1])*0.5;
					velocity[nx*N+ny][2]=velocity[nx*N+ny][2]+dt*(zForce[nx*N+ny] + oldforce[nx*N+ny][2])*0.5;
					//nodes[nx*N+ny].velocity+=dt*(vector(TotalForceEachDim[nx*N+ny][0],TotalForceEachDim[nx*N+ny][1],TotalForceEachDim[nx*N+ny][2])+nodes[nx*N+ny].oldforce)*0.5
				}
			}
			
			//for noting the time diffrence before and after parallelizing ..just for data gathering purpose..
			if(iter==1500)
			{
				clock_t clktime2 = clock();
				time_t RealTime2 = time(0);

				double diffClock = ((double)(clktime2-clktime1))/CLOCKS_PER_SEC;
				double diffSystem= difftime(RealTime2,RealTime1);					
				printf("CPU time:%f \n",diffClock);
				printf("Wall time:%f \n",diffSystem);
				printf("Force is %E %E %E\n",xForce[0],yForce[0],zForce[0]);
			}
		}
}

static PyMethodDef ClothEngineCUDA_methods[] =
{	
	{"cFullEngine",py_cFullEngine,METH_VARARGS},
	{NULL,NULL} /* Sentinel */
};

PyMODINIT_FUNC
	initClothEngineCUDA(){
		(void) Py_InitModule("ClothEngineCUDA",ClothEngineCUDA_methods);
}
