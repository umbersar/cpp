
#include <hip/hip_runtime.h>
//#include "Python.h"
//
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include "arrayobject.h"
//#include <math.h>
//#include <string.h>
//#include <stdio.h>
//
//
//#define IDATA(p) ((int *) (((PyArrayObject *)p)->data))
//#define DDATA(p) ((double *) (((PyArrayObject *)p)->data))
//
//static PyObject *py_cComputeForce(PyObject *self, PyObject *args);
//PyObject* c_compute_force(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
//	double *xPos, double *yPos, double *zPos);
//PyObject* c_compute_force_Cudafied(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
//	double *xPos, double *yPos, double *zPos);
//
//
//__device__ int maxOnDevice(int a, int b) {
//	if (a > b) {
//		return a;
//	} else {
//		return b;
//	}
//}//end max
//
//__device__ int minOnDevice(int a, int b) {
//	if (a > b) {
//		return b;
//	} else {
//		return a;
//	}
//}//end min
//
//__device__ double magOnDevice(double x, double y, double z) {
//	return sqrt(x*x + y*y +z*z);
//}//end min
//
//__device__ double normxOnDevice(double x, double y, double z) {
//	return x/sqrt(x*x + y*y +z*z);
//}//end min
//
//__device__ double normyOnDevice(double x, double y, double z) {
//	return y/sqrt(x*x + y*y +z*z);
//}//end min
//
//__device__ double normzOnDevice(double x, double y, double z) {
//	return z/sqrt(x*x + y*y +z*z);
//}//end min
//
//int maxOnHost(int a, int b) {
//	if (a > b) {
//		return a;
//	} else {
//		return b;
//	}
//}//end max
//
//int minOnHost(int a, int b) {
//	if (a > b) {
//		return b;
//	} else {
//		return a;
//	}
//}//end min
//
//double mag(double x, double y, double z) {
//	return sqrt(x*x + y*y +z*z);
//}//end min
//
//double normx(double x, double y, double z) {
//	return x/sqrt(x*x + y*y +z*z);
//}//end min
//
//double normy(double x, double y, double z) {
//	return y/sqrt(x*x + y*y +z*z);
//}//end min
//
//double normz(double x, double y, double z) {
//	return z/sqrt(x*x + y*y +z*z);
//}//end min
//
//static PyObject *py_cComputeForce(PyObject *self, PyObject *args){
//	int N,delta,ok;
//	double gravity , separation, fcon;
//
//	PyObject *xposarray, *yposarray, *zposarray;
//	PyObject *xforcearray, *yforcearray, *zforcearray;
//	double *xPos, *yPos, *zPos;
//	double *xForce, *yForce, *zForce;
//
//	//double pe;
//	PyObject *lst;
//
//	ok = PyArg_ParseTuple(args,"iidddOOOOOO",&N,&delta,&gravity,&separation,&fcon,&xforcearray,&yforcearray,&zforcearray,
//		&xposarray,&yposarray,&zposarray);
//
//	/*if (true){
//	fprintf(stdout,"N= %d, delta=%d gravity=%f, separation=%f fcon=%f \n",N,delta,gravity,separation,fcon);
//	exit(1);
//	}*/
//
//	if (!ok){
//		fprintf(stderr,"Error (cComputeForce) in parsing arguments\n");
//		exit(1);
//	}
//
//	xPos = DDATA(xposarray);
//	yPos = DDATA(yposarray);
//	zPos = DDATA(zposarray);
//	xForce = DDATA(xforcearray);
//	yForce = DDATA(yforcearray);
//	zForce = DDATA(zforcearray);
//
//	//pe = c_compute_force(N,delta,gravity,separation,fcon,xForce,yForce,zForce,xPos,yPos,zPos);
//	
//	lst = c_compute_force(N,delta,gravity,separation,fcon,xForce,yForce,zForce,xPos,yPos,zPos);
//	//lst = c_compute_force_Cudafied(N,delta,gravity,separation,fcon,xForce,yForce,zForce,xPos,yPos,zPos);
//
//	return lst;
//	//return Py_BuildValue("d",pe);
//}
//
////ramneek: trying to cuda'fy this code
//__global__ void MyKernel(int *Nptr,int *deltaptr, double *gravityptr, double *separationptr, double *fconptr, double *xForce, double *yForce, double *zForce,
//	double *xPos, double *yPos, double *zPos/*, PyObject *force_on_each_ball_list*/ )
//{
//	int N = *Nptr;
//	int delta= *deltaptr;
//	double gravity= *gravityptr;
//	double separation = *separationptr;
//	double fcon = *fconptr;
//
//	double len=0.0;
//	double r12X =0.0;
//	double r12Y =0.0;
//	double r12Z =0.0;
//	double PE=0.0;
//
//	
//	int nx = blockDim.x * blockIdx.x + threadIdx.x;//use this place of nx
//	int ny = blockDim.x * blockIdx.x + threadIdx.y;//use this place of ny
//	if(!(nx< N && ny <N))
//		return;
//
//	xForce[nx*N+ny] = 0.0;
//	yForce[nx*N+ny] = -gravity;
//	zForce[nx*N+ny] = 0.0;
//
//	int lowerValuedx = maxOnDevice(nx-delta,0);
//	int upperValuedx=minOnDevice(nx+delta+1,N);
//	for(int dx=lowerValuedx; dx<upperValuedx;dx++)
//	{
//		int lowerValuedy=maxOnDevice(ny-delta,0);
//		int upperValuedy=minOnDevice(ny+delta+1,N);
//		for(int dy=lowerValuedy; dy<upperValuedy;dy++)
//		{
//			len=sqrt((double)((nx-dx)*(nx-dx)+(ny-dy)*(ny-dy)) ) *separation;
//
//			bool condition = ny!=dy;
//			bool condition1 = nx!=dx;
//			if (condition || condition1)
//			//if (nx!=dx || ny!=dy)
//			{
//				r12X = xPos[dx*N+dy] - xPos[nx*N+ny];
//				r12Y = yPos[dx*N+dy] - yPos[nx*N+ny];
//				r12Z = zPos[dx*N+dy] - zPos[nx*N+ny];
//				PE = PE + fcon*(magOnDevice(r12X,r12Y,r12Z)-len)*(magOnDevice(r12X,r12Y,r12Z)-len);
//				xForce[nx*N+ny] = xForce[nx*N+ny] +fcon*normxOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);
//				yForce[nx*N+ny]= yForce[nx*N+ny] +fcon*normyOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);
//				zForce[nx*N+ny]= zForce[nx*N+ny] +fcon*normzOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);
//
//				//i tried to first get the item and modify it and set it back.
//				//but then i thought....why not directly set the new item that position if it is anyways going to overwrite it.
//				//PyObject *temp=PyList_GetItem(force_on_each_ball_list, nx*N+ny);
//				/*ok = PyArg_ParseTuple(temp,"ddd",&N,&delta,&gravity,&separation,&fcon,&xforcearray,&yforcearray,&zforcearray,
//				&xposarray,&yposarray,&zposarray);*/
//
//				//ramneek: get the items out of xForce, yForce and zForce in the host method and use the follwing statements there. 
//				/*PyObject *item = Py_BuildValue("(ddd)",xForce[nx*N+ny],yForce[nx*N+ny],zForce[nx*N+ny]);
//				PyList_SetItem(force_on_each_ball_list, nx*N+ny, item);*/
//			}
//		}
//	}
//	/*   int i = threadIdx.x;
//    c[i] = a[i] + b[i];*/
//}
//
//PyObject* c_compute_force_Cudafied(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
//	double *xPos, double *yPos, double *zPos){
//
//		double r12X =0.0;
//		double r12Y =0.0;
//		double r12Z =0.0;
//		//r12=vector(0.0,0.0,0.0)
//		int nx,ny,dx,dy;
//		double PE=0.0;
//		double len=0.0;
//
//
//		//now i m building the force afresh every time i call the method. So no need to pass the force arguments array to it.
//		//just return a force array(list) to python form here for it to consume it.
//		//now issue is force is list in python. i can make a list from here and return it.
//		//but i have to let some values remiain (0,0,0) in the list and some have to have force (fx,fy,fz).
//		//what i could do was make a list here.add (0,0,0) to it for every element by using PyList_Append.
//		//but when i run i main loop, i have to update the (fx,fy,fz) values for which interaction is taking place.
//		//so i have to get a element out of the list and update it. so find a method for updating which shoudl be similiar to PyList_Append.
//
//
//		PyObject *force_on_each_ball_list = Py_BuildValue("[]");
//		if (!force_on_each_ball_list)
//			return NULL;
//
//		for (nx=0; nx<N; nx++)
//		{
//			for (ny=0; ny<N; ny++)
//			{
//				PyObject *lc = Py_BuildValue("(ddd)",0.0,0.0,0.0);
//				PyList_Append(force_on_each_ball_list,lc);
//				Py_DECREF(lc);
//			}
//		}
//
//		//allocate memory on device here
//		int *dev_N =0;
//		int *dev_delta=0;
//		double *dev_gravity=0;
//		double *dev_separation=0;
//		double *dev_fcon=0;
//
//		double *dev_xForce = 0;
//		double *dev_yForce = 0;
//		double *dev_zForce = 0;
//		double *dev_xPos = 0;
//		double *dev_yPos = 0;
//		double *dev_zPos = 0;
//
//		cudaError_t cudaStatus;
//
//		// Choose which GPU to run on, change this on a multi-GPU system.
//		cudaStatus = cudaSetDevice(0);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//			exit(0);
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_N, sizeof(int));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_delta, sizeof(int));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_gravity, sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_separation, sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_fcon, sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		// Allocate GPU buffers for 6 vectors    .
//		cudaStatus = cudaMalloc((void**)&dev_xForce, N*N * sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_yForce,  N*N * sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_zForce,  N*N * sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//		cudaStatus = cudaMalloc((void**)&dev_xPos, N*N * sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_yPos,  N*N * sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMalloc((void**)&dev_zPos,  N*N * sizeof(double));
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMalloc failed!");
//			goto Error;
//		}
//
//
//		cudaStatus = cudaMemcpy(dev_N, &N,  sizeof(int), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_delta, &delta,  sizeof(int), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_gravity, &gravity,  sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_separation, &separation,  sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_fcon, &fcon,  sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		// Copy input vectors from host memory to GPU buffers.
//		cudaStatus = cudaMemcpy(dev_xForce, xForce, N*N * sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_yForce, yForce, N*N * sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_zForce, zForce, N*N * sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_xPos, xPos, N*N * sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_yPos, yPos, N*N * sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		cudaStatus = cudaMemcpy(dev_zPos, zPos, N*N * sizeof(double), cudaMemcpyHostToDevice);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		dim3 threadsPerBlock(16, 16); 
//		int blocksPerGrid = (N*N + (threadsPerBlock.x*threadsPerBlock.y) - 1) / (threadsPerBlock.x*threadsPerBlock.y);
//		MyKernel<<<blocksPerGrid, threadsPerBlock>>>( dev_N,dev_delta,dev_gravity,dev_separation,dev_fcon,dev_xForce,
//			dev_yForce,dev_zForce,dev_xPos,dev_yPos,dev_zPos); 
//
//		////launch the kernel here
//		//int numBlocks = 1; 
//		//dim3 threadsPerBlock(N, N); 
//		//MyKernel<<<numBlocks, threadsPerBlock>>>( dev_N,dev_delta,dev_gravity,dev_separation,dev_fcon,dev_xForce,
//		//	dev_yForce,dev_zForce,dev_xPos,dev_yPos,dev_zPos); 
//
//		// cudaDeviceSynchronize waits for the kernel to finish, and returns
//		// any errors encountered during the launch.
//		cudaStatus = cudaDeviceSynchronize();
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//			goto Error;
//		}
//
//		// Copy output vector from GPU buffer to host memory.
//		cudaStatus = cudaMemcpy(xForce, dev_xForce, N*N * sizeof(double), cudaMemcpyDeviceToHost);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//		// Copy output vector from GPU buffer to host memory.
//		cudaStatus = cudaMemcpy(yForce, dev_yForce, N*N * sizeof(double), cudaMemcpyDeviceToHost);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//		// Copy output vector from GPU buffer to host memory.
//		cudaStatus = cudaMemcpy(zForce, dev_zForce, N*N * sizeof(double), cudaMemcpyDeviceToHost);
//		if (cudaStatus != cudaSuccess) {
//			fprintf(stderr, "cudaMemcpy failed!");
//			goto Error;
//		}
//
//		//populate the list here
//		for (nx=0; nx<N; nx++)
//		{
//			for (ny=0; ny<N; ny++)
//			{
//				int lowerValuedx = maxOnHost(nx-delta,0);
//				int upperValuedx=minOnHost(nx+delta+1,N);
//				for(dx=lowerValuedx; dx<upperValuedx;dx++)
//				{
//					int lowerValuedy=maxOnHost(ny-delta,0);
//					int upperValuedy=minOnHost(ny+delta+1,N);
//					for(dy=lowerValuedy; dy<upperValuedy;dy++)
//					{
//						if (nx!=dx || ny!=dy)
//						{
//							PyObject *item = Py_BuildValue("(ddd)",xForce[nx*N+ny],yForce[nx*N+ny],zForce[nx*N+ny]);
//							PyList_SetItem(force_on_each_ball_list, nx*N+ny, item);
//						}
//					}
//				}
//			}
//		}
//		
//Error:
//		cudaFree(dev_N);
//		cudaFree(dev_delta);
//		cudaFree(dev_gravity);
//		cudaFree(dev_separation);
//		cudaFree(dev_fcon);
//
//		cudaFree(dev_xForce);
//		cudaFree(dev_yForce);
//		cudaFree(dev_zForce);
//		cudaFree(dev_xPos);
//		cudaFree(dev_yPos);
//		cudaFree(dev_zPos);
//
//		//pe returned is correct. so i m not returning it now. instead i will return a list of with each element being (xForce,yForce,zForce)
//		//return PE;
//		return force_on_each_ball_list;
//}
//
////ramneek:end
//
////todo: now remove all these parameters as i dont need them
//PyObject* c_compute_force(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
//	double *xPos, double *yPos, double *zPos){
//
//		double r12X =0.0;
//		double r12Y =0.0;
//		double r12Z =0.0;
//		//r12=vector(0.0,0.0,0.0)
//		int nx,ny,dx,dy;
//		double PE=0.0;
//		double len=0.0;
//
//
//		//now i m building the force afresh every time i call the method. So no need to pass the force arguments array to it.
//		//just return a force array(list) to python form here for it to consume it.
//		//now issue is force is list in python. i can make a list from here and return it.
//		//but i have to let some values remiain (0,0,0) in the list and some have to have force (fx,fy,fz).
//		//what i could do was make a list here.add (0,0,0) to it for every element by using PyList_Append.
//		//but when i run i main loop, i have to update the (fx,fy,fz) values for which interaction is taking place.
//		//so i have to get a element out of the list and update it. so find a method for updating which shoudl be similiar to PyList_Append.
//
//
//		PyObject *force_on_each_ball_list = Py_BuildValue("[]");
//		if (!force_on_each_ball_list)
//			return NULL;
//
//		for (nx=0; nx<N; nx++)
//		{
//			for (ny=0; ny<N; ny++)
//			{
//				PyObject *lc = Py_BuildValue("(ddd)",0.0,0.0,0.0);
//				PyList_Append(force_on_each_ball_list,lc);
//				Py_DECREF(lc);
//			}
//		}
//
//		for (nx=0; nx<N; nx++)
//		{
//			for (ny=0; ny<N; ny++)
//			{
//				xForce[nx*N+ny] = 0.0;
//				yForce[nx*N+ny] = -gravity;
//				zForce[nx*N+ny] = 0.0;
//
//				int lowerValuedx = max(nx-delta,0);
//				int upperValuedx=min(nx+delta+1,N);
//				for(dx=lowerValuedx; dx<upperValuedx;dx++)
//				{
//					int lowerValuedy=max(ny-delta,0);
//					int upperValuedy=min(ny+delta+1,N);
//					for(dy=lowerValuedy; dy<upperValuedy;dy++)
//					{
//						len=sqrt((double)((nx-dx)*(nx-dx)+(ny-dy)*(ny-dy)) ) *separation;
//
//						if (nx!=dx || ny!=dy)
//						{
//							r12X = xPos[dx*N+dy] - xPos[nx*N+ny];
//							r12Y = yPos[dx*N+dy] - yPos[nx*N+ny];
//							r12Z = zPos[dx*N+dy] - zPos[nx*N+ny];
//							PE = PE + fcon*(mag(r12X,r12Y,r12Z)-len)*(mag(r12X,r12Y,r12Z)-len);
//							xForce[nx*N+ny] = xForce[nx*N+ny] +fcon*normx(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
//							yForce[nx*N+ny]= yForce[nx*N+ny] +fcon*normy(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
//							zForce[nx*N+ny]= zForce[nx*N+ny] +fcon*normz(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
//
//							//i tried to first get the item and modify it and set it back.
//							//but then i thought....why not directly set the new item that position if it is anyways going to overwrite it.
//							//PyObject *temp=PyList_GetItem(force_on_each_ball_list, nx*N+ny);
//							/*ok = PyArg_ParseTuple(temp,"ddd",&N,&delta,&gravity,&separation,&fcon,&xforcearray,&yforcearray,&zforcearray,
//							&xposarray,&yposarray,&zposarray);*/
//
//							PyObject *item = Py_BuildValue("(ddd)",xForce[nx*N+ny],yForce[nx*N+ny],zForce[nx*N+ny]);
//							PyList_SetItem(force_on_each_ball_list, nx*N+ny, item);
//						}
//					}
//				}
//
//			}
//		}
//		//pe returned is correct. so i m not returning it now. instead i will return a list of with each element being (xForce,yForce,zForce)
//		//return PE;
//		return force_on_each_ball_list;
//}
//
//static PyMethodDef ClothEngine_methods[] =
//{	
//	{"cComputeForce",py_cComputeForce,METH_VARARGS},
//	{NULL,NULL} /* Sentinel */
//};
//
//PyMODINIT_FUNC
//	initClothEngine(){
//		(void) Py_InitModule("ClothEngine",ClothEngine_methods);
//}
