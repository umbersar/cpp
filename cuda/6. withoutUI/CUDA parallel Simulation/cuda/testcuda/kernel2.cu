#include "hip/hip_runtime.h"
//ramneek: after u have gone through the tute for openmp, try to make cloth engine as a stand alone component and then parralelize it
//#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <time.h>

void GetPosInXYZDirection(double** nodepos, double *xPos, double *yPos, double *zPos, int N);
void GetForceInXYZDirection(double** force, double *xForce, double *yForce, double *zForce, int N);
void GetVelInXYZDirection(double** velocity, double *xVel, double *yVel, double *zVel, int N);
void create_cloth(int N,double separation, double offset, double ballsize, double** velocity, double** force, double** oldforce, double** nodepos);
void c_compute_force(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
double *xPos, double *yPos, double *zPos);
void c_compute_force_Cudafied(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
double *xPos, double *yPos, double *zPos);
double mag(double x, double y, double z);

int main(int argc, char* argv[])
{
	//int N=3,np=1;
	int N=50,np=1;

	
	if (argc != 3) {
		printf(" %s Number_of_nodes Number_of_threads \n",argv[0]);
		//return -1;
	 }
	 else {
		N = atoi(argv[1]);
		np = atoi(argv[2]);
		if (N < 1){
		  printf("Error: Number_of_nodes (%i) < 1 \n",N);
		  return -1;
		}
	 }

	double myBallX = 0.0;
	double myBallY = 0.0;
	double myBallZ = 0.0;
	double myBallRadius = 3.0;

	double separation=1.0;
	double mass = 1.0;
	double fcon = 10.0;
	int interact = 2;
	double gravity = 1.0;
	double ballsize = 3.0;//this should match with myballRadius
	double offset = 0.0;
	double dt = 0.01;
	double update = 1;

	omp_set_num_threads(np);
	
	//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
	double **velocity;
	velocity  = (double **)malloc(N * N * sizeof(double *));
	for(int i = 0; i < N * N ; i++)
	velocity[i] = (double *)malloc(3 * sizeof(double));

	//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
	double **force;
	force  = (double **)malloc(N * N * sizeof(double *));
	for(int i = 0; i < N * N ; i++)
	force[i] = (double *)malloc(3 * sizeof(double));

	//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
	double **oldforce;
	oldforce  = (double **)malloc(N * N * sizeof(double *));
	for(int i = 0; i < N * N ; i++)
	oldforce[i] = (double *)malloc(3 * sizeof(double));

	//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
	double **nodepos;
	nodepos  = (double **)malloc(N * N * sizeof(double *));
	for(int i = 0; i < N * N ; i++)
	nodepos[i] = (double *)malloc(3 * sizeof(double));

	create_cloth(N,separation,offset,ballsize,velocity,force,oldforce,nodepos);

	double *xVel, *yVel,  *zVel;
	xVel = (double *)malloc(N*N*sizeof(double));
	yVel = (double *)malloc(N*N*sizeof(double));
	zVel = (double *)malloc(N*N*sizeof(double));
	GetVelInXYZDirection(velocity,xVel,yVel,zVel,N);
	//xPos, yPos, zPos = GetPosInXYZDirection(nodes)

	double *xPos, *yPos,  *zPos;
	xPos = (double *)malloc(N*N*sizeof(double));
	yPos = (double *)malloc(N*N*sizeof(double));
	zPos = (double *)malloc(N*N*sizeof(double));
	GetPosInXYZDirection(nodepos,xPos,yPos,zPos,N);
	//xPos, yPos, zPos = GetPosInXYZDirection(nodes)

	double *xForce, *yForce,  *zForce;
	xForce = (double *)malloc(N*N*sizeof(double));
	yForce = (double *)malloc(N*N*sizeof(double));
	zForce = (double *)malloc(N*N*sizeof(double));
	GetForceInXYZDirection(force,xForce,yForce,zForce,N);
	//xForce, yForce, zForce = GetForceInXYZDirection(nodes) 

	//c_compute_force(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);
	c_compute_force_Cudafied(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);

	clock_t clktime1 = clock();
	time_t RealTime1 = time(0);
	
	int iter=0;
	//while(iter<0)
	while(iter<3000)
	{
		iter=iter+1;

		for(int nx=0;nx<N;nx++)
		{
			for(int ny=0;ny<N;ny++)
			{
				nodepos[nx*N+ny][0] = nodepos[nx*N+ny][0] + dt*(velocity[nx*N+ny][0]+dt*xForce[nx*N+ny]*.5);
				nodepos[nx*N+ny][1] = nodepos[nx*N+ny][1] + dt*(velocity[nx*N+ny][1]+dt*yForce[nx*N+ny]*.5);
				nodepos[nx*N+ny][2] = nodepos[nx*N+ny][2] + dt*(velocity[nx*N+ny][2]+dt*zForce[nx*N+ny]*.5);

				oldforce[nx*N+ny][0] = xForce[nx*N+ny];
				oldforce[nx*N+ny][1] = yForce[nx*N+ny];
				oldforce[nx*N+ny][2] = zForce[nx*N+ny];

				// if(iter==300)
				// {
					// printf("%E %E %E\n",nodepos[nx*N+ny][0],nodepos[nx*N+ny][1],nodepos[nx*N+ny][2]);
					// // printf("%E %E %E\n",xForce[nx*N+ny],yForce[nx*N+ny],zForce[nx*N+ny]);
				// }
			}
		}
		
		for(int i=0;i<N*N;i++)
		{
			double distX = nodepos[i][0] - myBallX;
			double distY = nodepos[i][1] - myBallY;
			double distZ = nodepos[i][2] - myBallZ;

			double dist = mag(distX,distY,distZ);
			//dist = node.pos-vector(myball.x,myball.y,myball.z)

			if(dist<myBallRadius)
			{
				// printf("%E %E %E\n",nodepos[i][0] ,nodepos[i][1],nodepos[i][2]);
				double fvectorX = (distX/dist)*myBallRadius;
				double fvectorY = (distY/dist)*myBallRadius;
				double fvectorZ = (distZ/dist)*myBallRadius;
				//fvector=dist/dist.mag*myball.radius

				nodepos[i][0] = myBallX +fvectorX;
				nodepos[i][1] = myBallY +fvectorY;
				nodepos[i][2] = myBallZ +fvectorZ;
				//node.pos=vector(myball.x,myball.y,myball.z)+fvector	

				double fvectorMag = mag(fvectorX,fvectorY,fvectorZ);
				velocity[i][0] = velocity[i][0] - (velocity[i][0]*fvectorX/fvectorMag)*(fvectorX/fvectorMag);
				velocity[i][1] = velocity[i][1] - (velocity[i][1]*fvectorY/fvectorMag)*(fvectorY/fvectorMag);
				velocity[i][2] = velocity[i][2] - (velocity[i][2]*fvectorZ/fvectorMag)*(fvectorZ/fvectorMag);
				
				// printf("%E %E %E\n",fvectorX,fvectorY,fvectorZ);
				//node.velocity = node.velocity - (dot(node.velocity,fvector/fvector.mag))*(fvector/fvector.mag)
			}
		}
		
		GetPosInXYZDirection(nodepos,xPos,yPos,zPos,N);
		GetForceInXYZDirection(force,xForce,yForce,zForce,N);		
		//c_compute_force(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);
		c_compute_force_Cudafied(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);

		for(int nx=0;nx<N;nx++)
		{
			for(int ny=0;ny<N;ny++)
			{
				velocity[nx*N+ny][0]=velocity[nx*N+ny][0]+dt*(xForce[nx*N+ny] + oldforce[nx*N+ny][0])*0.5;
				velocity[nx*N+ny][1]=velocity[nx*N+ny][1]+dt*(yForce[nx*N+ny] + oldforce[nx*N+ny][1])*0.5;
				velocity[nx*N+ny][2]=velocity[nx*N+ny][2]+dt*(zForce[nx*N+ny] + oldforce[nx*N+ny][2])*0.5;
				//nodes[nx*N+ny].velocity+=dt*(vector(TotalForceEachDim[nx*N+ny][0],TotalForceEachDim[nx*N+ny][1],TotalForceEachDim[nx*N+ny][2])+nodes[nx*N+ny].oldforce)*0.5
				// if(iter==300)
				// {
				// printf("%E %E %E \n",velocity[nx*N+ny][0],velocity[nx*N+ny][1],velocity[nx*N+ny][2]);
				// }
			}
		}
		if(iter==2000)
		{
			clock_t clktime2 = clock();
			time_t RealTime2 = time(0);

			double diffClock = ((double)(clktime2-clktime1))/CLOCKS_PER_SEC;
			double diffSystem= difftime(RealTime2,RealTime1);					
			printf("CPU clock timetime:%f \n",diffClock);
			printf("Wall time:%f \n",diffSystem);
			printf("Force is %E %E %E\n",xForce[0],yForce[0],zForce[0]);
		}
	}
	getchar();
}

void GetPosInXYZDirection(double** nodepos, double *xPos, double *yPos, double *zPos, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
	xPos[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
	yPos[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
	zPos[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xPos[i] = nodepos[i][0];
		yPos[i] = nodepos[i][1];
		zPos[i] = nodepos[i][2];
	}
}

void GetForceInXYZDirection(double** force, double *xForce, double *yForce, double *zForce, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
	xForce[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
	yForce[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
	zForce[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xForce[i] = force[i][0];
		yForce[i] = force[i][1];
		zForce[i] = force[i][2];
	}
}

void GetVelInXYZDirection(double** velocity, double *xVel, double *yVel, double *zVel, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
	xVel[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
	yVel[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
	zVel[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xVel[i] = velocity[i][0];
		yVel[i] = velocity[i][1];
		zVel[i] = velocity[i][2];
	}
}

void create_cloth(int N,double separation, double offset, double ballsize, double** velocity, double** force, double** oldforce, double** nodepos)
{		
	//This is the conventional way to access two dim array. 
	//initialize the two dim array(matrix) to 
	// for(int i = 0; i < N*N; i++)
	// for(int j = 0; j < 3; j++)
	// velocity[i][j] = 0.0;

	//this is same but bit absurd way of doind same.
	// for(int nx=0;i<N;i++)
	// {
	// for(int ny=0;i<N;i++)
	// {
	// velocity[nx*N+ny][0] = 0.0;
	// }
	// }

	for(int nx=0;nx<N;nx++)
	{
		double x = nx*separation-(N-1)*separation*0.5+offset;
		for(int ny=0;ny<N;ny++)
		{
			double y = ny*separation-(N-1)*separation*0.5+offset;

			nodepos[nx*N+ny][0] = x;
			nodepos[nx*N+ny][1] = ballsize+1.0;
			nodepos[nx*N+ny][2] = y;

			velocity[nx*N+ny][0] = 0.0;
			velocity[nx*N+ny][1] = 0.0;
			velocity[nx*N+ny][2] = 0.0;

			force[nx*N+ny][0] = 0.0;
			force[nx*N+ny][1] = 0.0;
			force[nx*N+ny][2] = 0.0;

			oldforce[nx*N+ny][0] = 0.0;
			oldforce[nx*N+ny][1] = 0.0;
			oldforce[nx*N+ny][2] = 0.0;
		}
	}
}

__device__ int maxOnDevice(int a, int b) {
	if (a > b) {
		return a;
	} else {
		return b;
	}
}//end maxOnDevice

__device__ int minOnDevice(int a, int b) {
	if (a > b) {
		return b;
	} else {
		return a;
	}
}//end minOnDevice

__device__ double magOnDevice(double x, double y, double z) {
	return sqrt(x*x + y*y +z*z);
}//end minOnDevice

__device__ double normxOnDevice(double x, double y, double z) {
	return x/sqrt(x*x + y*y +z*z);
}//end minOnDevice

__device__ double normyOnDevice(double x, double y, double z) {
	return y/sqrt(x*x + y*y +z*z);
}//end minOnDevice

__device__ double normzOnDevice(double x, double y, double z) {
	return z/sqrt(x*x + y*y +z*z);
}//end minOnDevice

//int max(int a, int b) {
//	if (a > b) {
//		return a;
//	} else {
//		return b;
//	}
//}//end maxOnDevice
//
//int min(int a, int b) {
//	if (a > b) {
//		return b;
//	} else {
//		return a;
//	}
//}//end minOnDevice

double mag(double x, double y, double z) {
	return sqrt(x*x + y*y +z*z);
}//end minOnDevice

double normx(double x, double y, double z) {
	return x/sqrt(x*x + y*y +z*z);
}//end minOnDevice

double normy(double x, double y, double z) {
	return y/sqrt(x*x + y*y +z*z);
}//end minOnDevice

double normz(double x, double y, double z) {
	return z/sqrt(x*x + y*y +z*z);
}//end minOnDevice

//ramneek: trying to cuda'fy this code
__global__ void MyKernel(int *Nptr,int *deltaptr, double *gravityptr, double *separationptr, double *fconptr, double *xForce, double *yForce, double *zForce,
	double *xPos, double *yPos, double *zPos/*, PyObject *force_on_each_ball_list*/ )
{
	int N = *Nptr;
	//*Nptr =43332;
	int delta= *deltaptr;
	double gravity= *gravityptr;
	double separation = *separationptr;
	double fcon = *fconptr;

	double len=0.0;
	double r12X =0.0;
	double r12Y =0.0;
	double r12Z =0.0;
	double PE=0.0;


	int nx = blockDim.x * blockIdx.x + threadIdx.x;//use this place of nx
	//int ny = blockDim.x * blockIdx.x + threadIdx.y;//use this place of ny
	int ny = blockDim.y * blockIdx.y + threadIdx.y;
	//printf("nx:%d ny:%d\n", nx,ny);

	if(!(nx< N && ny <N))
		return;
	//printf("nx:%d ny:%d\n", nx,ny);


	xForce[nx*N+ny] = 0.0;
	yForce[nx*N+ny] = -gravity;
	zForce[nx*N+ny] = 0.0;

	int lowerValuedx = maxOnDevice(nx-delta,0);
	int upperValuedx=minOnDevice(nx+delta+1,N);
	for(int dx=lowerValuedx; dx<upperValuedx;dx++)
	{
		int lowerValuedy=maxOnDevice(ny-delta,0);
		int upperValuedy=minOnDevice(ny+delta+1,N);
		for(int dy=lowerValuedy; dy<upperValuedy;dy++)
		{
			len=sqrt((double)((nx-dx)*(nx-dx)+(ny-dy)*(ny-dy)) ) *separation;
			bool condition = ny!=dy;
			bool condition1 = nx!=dx;

			//if (nx!=dx || ny!=dy)
			if (condition || condition1)
			{
				r12X = xPos[dx*N+dy] - xPos[nx*N+ny];
				r12Y = yPos[dx*N+dy] - yPos[nx*N+ny];
				r12Z = zPos[dx*N+dy] - zPos[nx*N+ny];
				//PE = PE + fcon*(magOnDevice(r12X,r12Y,r12Z)-len)*(magOnDevice(r12X,r12Y,r12Z)-len);
				xForce[nx*N+ny] = xForce[nx*N+ny] +fcon*normxOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);
				yForce[nx*N+ny]= yForce[nx*N+ny] +fcon*normyOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);
				zForce[nx*N+ny]= zForce[nx*N+ny] +fcon*normzOnDevice(r12X,r12Y,r12Z)*(magOnDevice(r12X,r12Y,r12Z)-len);

				//i tried to first get the item and modify it and set it back.
				//but then i thought....why not directly set the new item that position if it is anyways going to overwrite it.
				//PyObject *temp=PyList_GetItem(force_on_each_ball_list, nx*N+ny);
				/*ok = PyArg_ParseTuple(temp,"ddd",&N,&delta,&gravity,&separation,&fcon,&xforcearray,&yforcearray,&zforcearray,
				&xposarray,&yposarray,&zposarray);*/

				//ramneek: get the items out of xForce, yForce and zForce in the host method and use the follwing statements there. 
				/*PyObject *item = Py_BuildValue("(ddd)",xForce[nx*N+ny],yForce[nx*N+ny],zForce[nx*N+ny]);
				PyList_SetItem(force_on_each_ball_list, nx*N+ny, item);*/
			}
		}
	}
	/*for (int k=0;k<256;k++)
	{
		xForce[k] = (double)444.7;
	}*/

	/*   int i = threadIdx.x;
    c[i] = a[i] + b[i];*/
}

void c_compute_force_Cudafied(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
	double *xPos, double *yPos, double *zPos){

		/*double r12X =0.0;
		double r12Y =0.0;
		double r12Z =0.0;*/
		//r12=vector(0.0,0.0,0.0)
		int nx,ny,dx,dy;
		/*double PE=0.0;
		double len=0.0;*/




		//allocate memory on device here
		int *dev_N =0;
		int *dev_delta=0;
		double *dev_gravity=0;
		double *dev_separation=0;
		double *dev_fcon=0;

		double *dev_xForce = 0;
		double *dev_yForce = 0;
		double *dev_zForce = 0;
		double *dev_xPos = 0;
		double *dev_yPos = 0;
		double *dev_zPos = 0;

		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			exit(0);
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_N, sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_delta, sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_gravity, sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_separation, sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_fcon, sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		// Allocate GPU buffers for 6 vectors    .
		cudaStatus = hipMalloc((void**)&dev_xForce, N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_yForce,  N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_zForce,  N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_xPos, N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_yPos,  N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_zPos,  N*N * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}


		cudaStatus = hipMemcpy(dev_N, &N,  sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_delta, &delta,  sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_gravity, &gravity,  sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_separation, &separation,  sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_fcon, &fcon,  sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_xForce, xForce, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_yForce, yForce, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_zForce, zForce, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_xPos, xPos, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_yPos, yPos, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_zPos, zPos, N*N * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		//launch the kernel here
		//dim3 threadsPerBlock(2, 2); 
		dim3 threadsPerBlock(16, 16); 
		//int blocksPerGrid = (N*N + (threadsPerBlock.x*threadsPerBlock.y) - 1) / (threadsPerBlock.x*threadsPerBlock.y);
		dim3 blocksPerGrid(ceil((double)N / threadsPerBlock.x), ceil((double)N / threadsPerBlock.y));
		// Launch a kernel on the GPU with one thread for each element.
		//addKernel<<<1, size>>>(dev_c, dev_a, dev_b,size);
		//addKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_c, dev_a, dev_b,N);
 		MyKernel<<<blocksPerGrid, threadsPerBlock>>>( dev_N,dev_delta,dev_gravity,dev_separation,dev_fcon,dev_xForce,
			dev_yForce,dev_zForce,dev_xPos,dev_yPos,dev_zPos); 

		/*int numBlocks = 1; 
		dim3 threadsPerBlock(N, N); */
		/*MyKernel<<<numBlocks, threadsPerBlock>>>( dev_N,dev_delta,dev_gravity,dev_separation,dev_fcon,dev_xForce,
			dev_yForce,dev_zForce,dev_xPos,dev_yPos,dev_zPos); */

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(xForce, dev_xForce, N*N * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		//int *testptr;
		//testptr = (int*)malloc(sizeof(int));
		//// Copy output vector from GPU buffer to host memory.
		//cudaStatus = hipMemcpy(testptr, dev_N, sizeof(int), hipMemcpyDeviceToHost);
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "hipMemcpy failed!");
		//	goto Error;
		//}

		/*for (int k=0;k<256;k++)
		{
			printf(" force %E",xForce[k] );
		}*/

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(yForce, dev_yForce, N*N * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(zForce, dev_zForce, N*N * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		
Error:
		hipFree(dev_N);
		hipFree(dev_delta);
		hipFree(dev_gravity);
		hipFree(dev_separation);
		hipFree(dev_fcon);

		hipFree(dev_xForce);
		hipFree(dev_yForce);
		hipFree(dev_zForce);
		hipFree(dev_xPos);
		hipFree(dev_yPos);
		hipFree(dev_zPos);
}

//todo: now remove all these parameters as i dont need them
void c_compute_force(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
double *xPos, double *yPos, double *zPos){

	// double r12X =0.0;
	// double r12Y =0.0;
	// double r12Z =0.0;
	//r12=vector(0.0,0.0,0.0)
	int nx,ny,dx,dy;
	//double PE=0.0;
	//double len=0.0;

	#pragma omp parallel for default(none) shared(N,delta,gravity,separation,fcon,xForce,yForce,zForce,xPos,yPos,zPos) private(nx,ny,dx,dy)
	for (nx=0; nx<N; nx++)
	{
		for (ny=0; ny<N; ny++)
		{
			xForce[nx*N+ny] = 0.0;
			yForce[nx*N+ny] = -gravity;
			zForce[nx*N+ny] = 0.0;

			int lowerValuedx = max(nx-delta,0);
			int upperValuedx=min(nx+delta+1,N);
			for(dx=lowerValuedx; dx<upperValuedx;dx++)
			{
				int lowerValuedy=max(ny-delta,0);
				int upperValuedy=min(ny+delta+1,N);
				for(dy=lowerValuedy; dy<upperValuedy;dy++)
				{
					double len=sqrt((double)((nx-dx)*(nx-dx)+(ny-dy)*(ny-dy)) ) *separation;

					if (nx!=dx || ny!=dy)
					{
						double r12X = xPos[dx*N+dy] - xPos[nx*N+ny];
						double r12Y = yPos[dx*N+dy] - yPos[nx*N+ny];
						double r12Z = zPos[dx*N+dy] - zPos[nx*N+ny];
						//PE = PE + fcon*(mag(r12X,r12Y,r12Z)-len)*(mag(r12X,r12Y,r12Z)-len);
						xForce[nx*N+ny] = xForce[nx*N+ny] +fcon*normx(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
						yForce[nx*N+ny]= yForce[nx*N+ny] +fcon*normy(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
						zForce[nx*N+ny]= zForce[nx*N+ny] +fcon*normz(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
					}
				}
			}

		}
	}
}