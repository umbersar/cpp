#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Python.h"
#include "arrayobject.h"
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <omp.h>

#define IDATA(p) ((int *) (((PyArrayObject *)p)->data))
#define DDATA(p) ((double *) (((PyArrayObject *)p)->data))

void c_compute_force(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
	double *xPos, double *yPos, double *zPos);

static PyObject *py_cFullEngine(PyObject *self, PyObject *args);
void c_full_engine(int N,int interact,double myBallX,double myBallY,double myBallZ, double myBallRadius, double separation, double mass,double fcon,
	double gravity,double ballsize,double offset,double dt,int update,  void (*renderui)(double** nodepos, int N), int threads);

void GetPosInXYZDirection(double** nodepos, double *xPos, double *yPos, double *zPos, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
		xPos[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		yPos[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		zPos[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xPos[i] = nodepos[i][0];
		yPos[i] = nodepos[i][1];
		zPos[i] = nodepos[i][2];
	}
}

void GetForceInXYZDirection(double** force, double *xForce, double *yForce, double *zForce, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
		xForce[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		yForce[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		zForce[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xForce[i] = force[i][0];
		yForce[i] = force[i][1];
		zForce[i] = force[i][2];
	}
}

void GetVelInXYZDirection(double** velocity, double *xVel, double *yVel, double *zVel, int N)
{
	//initialize the two dim array(matrix) to 
	for(int i = 0; i < N*N; i++)		
		xVel[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		yVel[i] = 0.0;
	for(int i = 0; i < N*N; i++)		
		zVel[i] = 0.0;

	for(int i = 0; i < N*N; i++)		
	{
		xVel[i] = velocity[i][0];
		yVel[i] = velocity[i][1];
		zVel[i] = velocity[i][2];
	}
}

//this function is similiar to create_cloth as it initializes the data structures used for simulation
void Initialize(int N,double separation, double offset, double ballsize, double** velocity, double** force, double** oldforce, double** nodepos)
{		
	//This is the conventional way to access two dim array. 
	//initialize the two dim array(matrix) to 
	// for(int i = 0; i < N*N; i++)
	// for(int j = 0; j < 3; j++)
	// velocity[i][j] = 0.0;

	//this is same but a bit absurd way of doing same.
	// for(int nx=0;i<N;i++)
	// {
	// for(int ny=0;i<N;i++)
	// {
	// velocity[nx*N+ny][0] = 0.0;
	// }
	// }

	for(int nx=0;nx<N;nx++)
	{
		double x = nx*separation-(N-1)*separation*0.5+offset;
		for(int ny=0;ny<N;ny++)
		{
			double y = ny*separation-(N-1)*separation*0.5+offset;

			nodepos[nx*N+ny][0] = x;
			nodepos[nx*N+ny][1] = ballsize+1.0;
			nodepos[nx*N+ny][2] = y;

			velocity[nx*N+ny][0] = 0.0;
			velocity[nx*N+ny][1] = 0.0;
			velocity[nx*N+ny][2] = 0.0;

			force[nx*N+ny][0] = 0.0;
			force[nx*N+ny][1] = 0.0;
			force[nx*N+ny][2] = 0.0;

			oldforce[nx*N+ny][0] = 0.0;
			oldforce[nx*N+ny][1] = 0.0;
			oldforce[nx*N+ny][2] = 0.0;
		}
	}
}

int max(int a, int b) {
	if (a > b) {
		return a;
	} else {
		return b;
	}
}//end max

int min(int a, int b) {
	if (a > b) {
		return b;
	} else {
		return a;
	}
}//end min

double mag(double x, double y, double z) {
	return sqrt(x*x + y*y +z*z);
}//end min

double normx(double x, double y, double z) {
	return x/sqrt(x*x + y*y +z*z);
}//end min

double normy(double x, double y, double z) {
	return y/sqrt(x*x + y*y +z*z);
}//end min

double normz(double x, double y, double z) {
	return z/sqrt(x*x + y*y +z*z);
}//end min

static PyObject *py_renderui_func = NULL;

static void stub_renderui_func(double** nodepos, int N)
{
	/*int testargument=121;
	PyObject *arglist = Py_BuildValue("(i)", testargument);*/
	PyObject *pos_of_each_ball_list = Py_BuildValue("[]");
	if (!pos_of_each_ball_list)
		printf("there is an error");
	for (int nx=0; nx<N; nx++)
	{
		for (int ny=0; ny<N; ny++)
		{
			PyObject *lc = Py_BuildValue("(ddd)",nodepos[nx*N+ny][0],nodepos[nx*N+ny][1],nodepos[nx*N+ny][2]);
			PyList_Append(pos_of_each_ball_list,lc);
			Py_DECREF(lc);
		}
	}
	PyObject *arglist = Py_BuildValue("(O)",pos_of_each_ball_list);
	
	// ...for calling the Python rendering function.
	PyObject *result = PyEval_CallObject(py_renderui_func,arglist);

	Py_DECREF(arglist);
}

void c_compute_force(int N,int delta, double gravity, double separation, double fcon, double *xForce, double *yForce, double *zForce,
double *xPos, double *yPos, double *zPos){

	// double r12X =0.0;
	// double r12Y =0.0;
	// double r12Z =0.0;
	//r12=vector(0.0,0.0,0.0)
	int nx,ny,dx,dy;
	//double PE=0.0;
	//double len=0.0;

	#pragma omp parallel for default(none) shared(N,delta,gravity,separation,fcon,xForce,yForce,zForce,xPos,yPos,zPos) private(nx,ny,dx,dy)	
	for (nx=0; nx<N; nx++)
	{
		for (ny=0; ny<N; ny++)
		{
			xForce[nx*N+ny] = 0.0;
			yForce[nx*N+ny] = -gravity;
			zForce[nx*N+ny] = 0.0;

			int lowerValuedx = max(nx-delta,0);
			int upperValuedx=min(nx+delta+1,N);
			for(dx=lowerValuedx; dx<upperValuedx;dx++)
			{
				int lowerValuedy=max(ny-delta,0);
				int upperValuedy=min(ny+delta+1,N);
				for(dy=lowerValuedy; dy<upperValuedy;dy++)
				{
					double len=sqrt((double)((nx-dx)*(nx-dx)+(ny-dy)*(ny-dy)) ) *separation;

					if (nx!=dx || ny!=dy)
					{
						double r12X = xPos[dx*N+dy] - xPos[nx*N+ny];
						double r12Y = yPos[dx*N+dy] - yPos[nx*N+ny];
						double r12Z = zPos[dx*N+dy] - zPos[nx*N+ny];
						//PE = PE + fcon*(mag(r12X,r12Y,r12Z)-len)*(mag(r12X,r12Y,r12Z)-len);
						xForce[nx*N+ny] = xForce[nx*N+ny] +fcon*normx(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
						yForce[nx*N+ny]= yForce[nx*N+ny] +fcon*normy(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
						zForce[nx*N+ny]= zForce[nx*N+ny] +fcon*normz(r12X,r12Y,r12Z)*(mag(r12X,r12Y,r12Z)-len);
					}
				}
			}

		}
	}
}

static PyObject *py_cFullEngine(PyObject *self, PyObject *args){
	int N,interact,update,ok,threads;
	double myBallX,myBallY,myBallZ,myBallRadius,separation,mass,fcon,gravity,ballsize,offset,dt;


	PyObject *pyrenderuiobj;

	//cFullEngine(N,interact,myBallX,myBallY,myBallZ,myBallRadius,separation,mass,fcon,gravity,ballsize,offset,dt,update)
	ok = PyArg_ParseTuple(args,"iidddddddddddiOi",&N,&interact,&myBallX,&myBallY,&myBallZ,&myBallRadius,&separation,&mass,&fcon,
		&gravity,&ballsize,&offset,&dt,&update,&pyrenderuiobj,&threads);

	// make sure second argument is a function
	if (!PyCallable_Check(pyrenderuiobj)) {
		PyErr_SetString(PyExc_TypeError, "Need a callable object!");
	}

	py_renderui_func = pyrenderuiobj;

	if (!ok){
		fprintf(stderr,"Error (cComputeForce) in parsing arguments\n");
		exit(1);
	}

	c_full_engine(N,interact,myBallX,myBallY,myBallZ,myBallRadius,separation,mass,fcon,gravity,ballsize,offset,
		dt,update,stub_renderui_func,threads);

	//i am returning this as i think the entry functions are required to have a return type.
	PyObject *lst;
	return lst;
}

void c_full_engine(int N,int interact,double myBallX,double myBallY,double myBallZ, double myBallRadius, double separation, double mass,double fcon,
	double gravity,double ballsize,double offset,double dt,int update,  void (*renderui)(double** nodepos, int N), int threads){
	
		//printf("N=%d interact=%d ballx=%f bally%f ballz%f ballradius%f separation=%f mass=%f fcon=%f gravity=%f ballsize=%f offfset=%f dt=%f update=%d\n",
			//N,interact,myBallX,myBallY,myBallZ, myBallRadius, separation, mass,fcon,gravity,ballsize,offset,dt,update);

		//set the number of threads desired for simulation
		printf("threads are= %d\n",threads);
		printf("no. of processors are= %d\n",omp_get_num_procs());
		omp_set_num_threads(threads);

		//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
		double **velocity;
		velocity  = (double **)malloc(N * N * sizeof(double *));
		for(int i = 0; i < N * N ; i++)
			velocity[i] = (double *)malloc(3 * sizeof(double));

		//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
		double **force;
		force  = (double **)malloc(N * N * sizeof(double *));
		for(int i = 0; i < N * N ; i++)
			force[i] = (double *)malloc(3 * sizeof(double));

		//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
		double **oldforce;
		oldforce  = (double **)malloc(N * N * sizeof(double *));
		for(int i = 0; i < N * N ; i++)
			oldforce[i] = (double *)malloc(3 * sizeof(double));

		//just for declaring it i will use a more conventional access pattern. otherwise i will access using nx*N+ny
		double **nodepos;
		nodepos  = (double **)malloc(N * N * sizeof(double *));
		for(int i = 0; i < N * N ; i++)
			nodepos[i] = (double *)malloc(3 * sizeof(double));

		Initialize(N,separation,offset,ballsize,velocity,force,oldforce,nodepos);

		double *xVel, *yVel,  *zVel;
		xVel = (double *)malloc(N*N*sizeof(double));
		yVel = (double *)malloc(N*N*sizeof(double));
		zVel = (double *)malloc(N*N*sizeof(double));
		GetVelInXYZDirection(velocity,xVel,yVel,zVel,N);
		//xPos, yPos, zPos = GetPosInXYZDirection(nodes)

		double *xPos, *yPos,  *zPos;
		xPos = (double *)malloc(N*N*sizeof(double));
		yPos = (double *)malloc(N*N*sizeof(double));
		zPos = (double *)malloc(N*N*sizeof(double));
		GetPosInXYZDirection(nodepos,xPos,yPos,zPos,N);
		//xPos, yPos, zPos = GetPosInXYZDirection(nodes)

		double *xForce, *yForce,  *zForce;
		xForce = (double *)malloc(N*N*sizeof(double));
		yForce = (double *)malloc(N*N*sizeof(double));
		zForce = (double *)malloc(N*N*sizeof(double));
		GetForceInXYZDirection(force,xForce,yForce,zForce,N);
		//xForce, yForce, zForce = GetForceInXYZDirection(nodes) 

		c_compute_force(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);

		clock_t clktime1 = clock();
		time_t RealTime1 = time(0); 

		int iter=0;
		while(1)
		{
			iter=iter+1;
			
			for(int nx=0;nx<N;nx++)
			{
				for(int ny=0;ny<N;ny++)
				{
					nodepos[nx*N+ny][0] = nodepos[nx*N+ny][0] + dt*(velocity[nx*N+ny][0]+dt*xForce[nx*N+ny]*.5);
					nodepos[nx*N+ny][1] = nodepos[nx*N+ny][1] + dt*(velocity[nx*N+ny][1]+dt*yForce[nx*N+ny]*.5);
					nodepos[nx*N+ny][2] = nodepos[nx*N+ny][2] + dt*(velocity[nx*N+ny][2]+dt*zForce[nx*N+ny]*.5);

					oldforce[nx*N+ny][0] = xForce[nx*N+ny];
					oldforce[nx*N+ny][1] = yForce[nx*N+ny];
					oldforce[nx*N+ny][2] = zForce[nx*N+ny];
				}
			}
		
			for(int i=0;i<N*N;i++)
			{
				double distX = nodepos[i][0] - myBallX;
				double distY = nodepos[i][1] - myBallY;
				double distZ = nodepos[i][2] - myBallZ;

				double dist = mag(distX,distY,distZ);
				//dist = node.pos-vector(myball.x,myball.y,myball.z)

				if(dist<myBallRadius)
				{
					// printf("%E %E %E\n",nodepos[i][0] ,nodepos[i][1],nodepos[i][2]);
					double fvectorX = (distX/dist)*myBallRadius;
					double fvectorY = (distY/dist)*myBallRadius;
					double fvectorZ = (distZ/dist)*myBallRadius;
					//fvector=dist/dist.mag*myball.radius

					nodepos[i][0] = myBallX +fvectorX;
					nodepos[i][1] = myBallY +fvectorY;
					nodepos[i][2] = myBallZ +fvectorZ;
					//node.pos=vector(myball.x,myball.y,myball.z)+fvector	

					double fvectorMag = mag(fvectorX,fvectorY,fvectorZ);
					velocity[i][0] = velocity[i][0] - (velocity[i][0]*fvectorX/fvectorMag)*(fvectorX/fvectorMag);
					velocity[i][1] = velocity[i][1] - (velocity[i][1]*fvectorY/fvectorMag)*(fvectorY/fvectorMag);
					velocity[i][2] = velocity[i][2] - (velocity[i][2]*fvectorZ/fvectorMag)*(fvectorZ/fvectorMag);
					//node.velocity = node.velocity - (dot(node.velocity,fvector/fvector.mag))*(fvector/fvector.mag)
				}
			}

			if(iter%update==0) 
			{
				renderui(nodepos,N);
			}

			GetPosInXYZDirection(nodepos,xPos,yPos,zPos,N);
			GetForceInXYZDirection(force,xForce,yForce,zForce,N);		
			c_compute_force(N,interact,gravity,separation,fcon,xForce, yForce, zForce,xPos, yPos, zPos);

			for(int nx=0;nx<N;nx++)
			{
				for(int ny=0;ny<N;ny++)
				{
					velocity[nx*N+ny][0]=velocity[nx*N+ny][0]+dt*(xForce[nx*N+ny] + oldforce[nx*N+ny][0])*0.5;
					velocity[nx*N+ny][1]=velocity[nx*N+ny][1]+dt*(yForce[nx*N+ny] + oldforce[nx*N+ny][1])*0.5;
					velocity[nx*N+ny][2]=velocity[nx*N+ny][2]+dt*(zForce[nx*N+ny] + oldforce[nx*N+ny][2])*0.5;
					//nodes[nx*N+ny].velocity+=dt*(vector(TotalForceEachDim[nx*N+ny][0],TotalForceEachDim[nx*N+ny][1],TotalForceEachDim[nx*N+ny][2])+nodes[nx*N+ny].oldforce)*0.5
				}
			}
			
			//for noting the time diffrence before and after parallelizing ..just for data gathering purpose..
			if(iter==1500)
			{
				clock_t clktime2 = clock();
				time_t RealTime2 = time(0);

				double diffClock = ((double)(clktime2-clktime1))/CLOCKS_PER_SEC;
				double diffSystem= difftime(RealTime2,RealTime1);					
				printf("CPU time:%f \n",diffClock);
				printf("Wall time:%f \n",diffSystem);
				printf("Force is %E %E %E\n",xForce[0],yForce[0],zForce[0]);
			}
		}
}

static PyMethodDef ClothEngineopenMP_methods[] =
{	
	{"cFullEngine",py_cFullEngine,METH_VARARGS},
	{NULL,NULL} /* Sentinel */
};

PyMODINIT_FUNC
	initClothEngineopenMP(){
		(void) Py_InitModule("ClothEngineopenMP",ClothEngineopenMP_methods);
}
